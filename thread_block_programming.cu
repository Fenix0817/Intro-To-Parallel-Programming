
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello()
{
    printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
    // It has 16! different ways in which the thread blocks can be run
}


int main(int argc,char **argv)
{
    // launch the kernel
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    // force the printf()s to flush
    hipDeviceSynchronize();

    printf("That's all!\n");

    return 0;
}